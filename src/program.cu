#include "kernels/mykernels.cuh"
#include "helpers/myhelpers.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <random>

/*
* Function to populate an array of floats with random values
*/
void populate_array(float *arr, int size, std::mt19937 &gen, std::uniform_real_distribution<float> &dis){
    for(int i=0; i<size; i++){
        arr[i] = dis(gen);
    }
}

/*
* Invocation starts here.
*/
int main(){
    int m = MMM;
    int n = MMN;
    int k = MMK;
    int sizeA = m * k;
    int sizeB = k * n;
    int sizeC = m * n;
    int sF = sizeof(float);
    int warmup_runs = 5;
    int measurement_runs = 50;
    int numoperations = m * n * 2 * k;
    float *h_A, *h_B, *h_C, *h_C_cublas;
    float *d_A, *d_B, *d_C;
    float elapsed_time, cublas_elapsed_time;
    hipEvent_t beg, end, cublasBeg, cublasEnd;
    void (*invoke_kernel)(float *, float *, float *, int, int, int);


    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(50.0, 25.0);
    
    h_A = new float[sizeA];
    h_B = new float[sizeB];
    h_C = new float[sizeC];
    h_C_cublas = new float[sizeC];
    hipMalloc(&d_A, sF * sizeA);
    hipMalloc(&d_B, sF * sizeB);
    hipMalloc(&d_C, sF * sizeC);

    populate_array(h_A, sizeA, gen, dis);
    populate_array(h_B, sizeB, gen, dis);
    hipMemcpy(d_A, h_A, sF*sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sF*sizeB, hipMemcpyHostToDevice);

    hipEventCreate(&beg);
    hipEventCreate(&end);
    hipEventCreate(&cublasBeg);
    hipEventCreate(&cublasEnd);
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    hipEventRecord(cublasBeg);
    invoke_cublas_kernel(d_A, d_B, d_C, m, k, n, handle);
    hipEventRecord(cublasEnd);
    hipEventSynchronize(cublasBeg);
    hipEventSynchronize(cublasEnd);
    hipEventElapsedTime(&cublas_elapsed_time, cublasBeg, cublasEnd);
    hipMemcpy(h_C_cublas, d_C, sF*sizeC, hipMemcpyDeviceToHost);

    invoke_kernel = invoke_naive_matmul;

    // Startup check
    invoke_kernel(d_A, d_B, d_C, m, k, n);
    hipMemcpy(h_C, d_C, sF*sizeC, hipMemcpyDeviceToHost);
    int randomRow = gen() % m;
    int randomCol = gen() % n;
    if(h_C[randomRow * n + randomCol] != h_C_cublas[randomCol * m + randomRow]){
        std::cout << "Error: Cublas and my kernel results do not match" << std::endl;
        return 1;
    }
    for(int i=0; i<warmup_runs-1; i++){
        invoke_kernel(d_A, d_B, d_C, m, k, n);
    }

    hipEventRecord(beg);
    for(int i=0; i<measurement_runs; i++){
        invoke_kernel(d_A, d_B, d_C, m, k, n);
    }
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);

    std::cout << std::fixed << std::setprecision(5);

    std::cout<<"Time taken by cublas kernel: "<<cublas_elapsed_time<<" ms"<<std::endl;
    std::cout<<"Cublas GFLOPS: "<<(numoperations / (cublas_elapsed_time / 1000)) / 1e9<<std::endl;
    std::cout<<"Time taken by my kernel: "<<elapsed_time/measurement_runs<<" ms"<<std::endl;
    std::cout<<"Kernel GFLOPS: "<<(numoperations / ((elapsed_time/measurement_runs) / 1000)) / 1e9<<std::endl;
    std::cout<<"Relative performance: "<<cublas_elapsed_time / (elapsed_time/measurement_runs)<<std::endl;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_cublas;
    hipblasDestroy(handle);
}

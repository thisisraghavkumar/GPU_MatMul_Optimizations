#include "hip/hip_runtime.h"
#include "kernels/mykernels.cuh"
#include "helpers/myhelpers.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <random>
#include <algorithm>

/*
void naive_mat_mul(float *A, float *B, float *C, int m, int k, int n){
	for(int i=0; i<m; i++){
		for(int j=0; j<n; j++){
			float sum = 0.0f;
			for(int l=0; l<k; l++){
				sum += A[i * k + l] * B[l * n + j];
			}
			C[i * n + j] = sum;
		}
	}
}
*/

/*
* Function to populate an array of floats with random values
*/
void populate_array(float *arr, int size, std::mt19937 &gen, std::uniform_real_distribution<float> &dis){
    for(int i=0; i<size; i++){
        arr[i] = dis(gen);
    }
}

/*
* Invocation starts here.
*/
int main(){
    int m = MMM;
    int n = MMN;
    int k = MMK;
    int sizeA = m * k;
    int sizeB = k * n;
    int sizeC = m * n;
    int sF = sizeof(float);
    int warmup_runs = 5;
    int measurement_runs = 50;
    int numoperations = m * n * 2 * k;
    float *h_A, *h_B, *h_C, *h_C_cublas;//, *h_C_ref;
    float *d_A, *d_B, *d_C;
    float elapsed_time, cublas_elapsed_time;
    hipEvent_t beg, end, cublasBeg, cublasEnd;
    void (*invoke_kernel)(float *, float *, float *, int, int, int);


    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(50.0, 25.0);
    
    h_A = new float[sizeA]();
    h_B = new float[sizeB]();
    h_C = new float[sizeC]();
    h_C_cublas = new float[sizeC]();
    //h_C_ref = new float[sizeC]();
    hipMalloc(&d_A, sF * sizeA);
    hipMalloc(&d_B, sF * sizeB);
    hipMalloc(&d_C, sF * sizeC);

    //float valuesA[] = {1.0f, 2.0f, 5.0f, 4.0f, 6.0f, 8.0f, 2.0f, 3.0f, 2.0f};
    populate_array(h_A, sizeA, gen, dis);
    //std::copy(valuesA,valuesA+9,h_A);
    //float valuesB[] = {1.0f, 0.0f, 2.0f, 2.0f, 1.0f, 1.0f, 8.0f, 2.0f, 4.0f};
    populate_array(h_B, sizeB, gen, dis);
    //std::copy(valuesB,valuesB+9,h_B);
    hipMemcpy(d_A, h_A, sF*sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sF*sizeB, hipMemcpyHostToDevice);
    //naive_mat_mul(h_A, h_B, h_C_ref, m, k, n);
    hipEventCreate(&beg);
    hipEventCreate(&end);
    hipEventCreate(&cublasBeg);
    hipEventCreate(&cublasEnd);
    

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    invoke_cublas_kernel(d_A, d_B, d_C, m, k, n, handle);
    hipDeviceSynchronize();
    hipMemcpy(h_C_cublas, d_C, sF*sizeC, hipMemcpyDeviceToHost);

    invoke_kernel = invoke_rowmajor_matmul;

    // Startup check
    invoke_kernel(d_A, d_B, d_C, m, k, n);
    hipMemcpy(h_C, d_C, sF*sizeC, hipMemcpyDeviceToHost);
    int randomRow = gen() % m;
    int randomCol = gen() % n;
    float tolerance = 1;
    if(fabs(h_C[randomRow * n + randomCol] - h_C_cublas[randomRow * n + randomCol]) > tolerance){
        std::cout << "Error: Cublas and my kernel results do not match at "<<randomRow<<", "<<randomCol << std::endl;
	std::cout <<"Content of h_C = "<<std::setprecision(32)<<h_C[randomRow * n + randomCol]<<std::endl;
	std::cout <<"Content of h_C_cublas = "<<std::setprecision(32)<<h_C_cublas[randomRow * n + randomCol]<<std::endl;

        return 1;
    }
    for(int i=0; i<warmup_runs-1; i++){
        invoke_kernel(d_A, d_B, d_C, m, k, n);
    }
    hipEventRecord(cublasBeg);
    for(int i=0; i<measurement_runs; i++){
        invoke_cublas_kernel(d_A, d_B, d_C, m, k, n,handle);
	hipDeviceSynchronize();
    }
    hipEventRecord(cublasEnd);
    hipEventSynchronize(cublasBeg);
    hipEventSynchronize(cublasEnd);
    hipEventElapsedTime(&cublas_elapsed_time, cublasBeg, cublasEnd);
 
    hipEventRecord(beg);
    for(int i=0; i<measurement_runs; i++){
        invoke_kernel(d_A, d_B, d_C, m, k, n);
	hipDeviceSynchronize();
    }
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed_time, beg, end);

    std::cout << std::fixed << std::setprecision(5);
    std::cout<<"Number of operations: "<<numoperations<<std::endl;
    std::cout<<"Time taken by cublas kernel: "<<cublas_elapsed_time/measurement_runs<<" ms"<<std::endl;
    std::cout<<"Cublas GFLOPS: "<<(numoperations / ((cublas_elapsed_time / measurement_runs) / 1000)) / 1e9<<std::endl;
    std::cout<<"Time taken by my kernel: "<<elapsed_time/measurement_runs<<" ms"<<std::endl;
    std::cout<<"Kernel GFLOPS: "<<(numoperations / ((elapsed_time/measurement_runs) / 1000)) / 1e9<<std::endl;
    std::cout<<"Relative performance: "<<cublas_elapsed_time / (elapsed_time/measurement_runs)<<std::endl;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_cublas;
    hipblasDestroy(handle);
}

#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <random>



#define BLOCK_SIZE 32
#define MMM 1024LL
#define MMK 1024LL
#define MMN 1024LL
#define sF sizeof(float)

#define CEIL_DIV(dividend, divisor) ((dividend + divisor - 1) / divisor)

void populate_array(float *arr, int size, std::mt19937 &gen, std::uniform_real_distribution<float> &dis)
{
    for (int i = 0; i < size; i++)
    {
        arr[i] = dis(gen);
    }
}

template <const uint BLOCKSIZE>
__global__ void sgemm_global_mem_coalesce(int M, int N, int K, float alpha,
                                          const float *A, const float *B,
                                          float beta, float *C) {
  const int cRow = blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
  const int cCol = blockIdx.y * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);

  // if statement is necessary to make things work under tile quantization
  if (cRow < M && cCol < N) {
    float tmp = 0.0;
    for (int i = 0; i < K; ++i) {
      tmp += A[cRow * K + i] * B[i * N + cCol];
    }
    C[cRow * N + cCol] = alpha * tmp + beta * C[cRow * N + cCol];
  }
}

void run_sgemm_coalesce(int M, int N, int K, float alpha, float *A, float *B,
                        float beta, float *C) {
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32 * 32);
  sgemm_global_mem_coalesce<32>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

template <const uint BLOCKSIZE>
__global__ void myRowCoalesceKernel(float *A, float *B, float *C, int m, int k, int n){
    int firstRow = blockIdx.x * BLOCKSIZE + threadIdx.x/BLOCKSIZE;
    int secondCol = blockIdx.y * BLOCKSIZE + threadIdx.x%BLOCKSIZE;

    if(firstRow < m && secondCol < n){
        float sum = 0.0f;
        for(int i=0; i<k; ++i){
            sum += A[firstRow * k + i] * B[i * n + secondCol];
        }
        C[firstRow * n + secondCol] = sum;
    }
}

void invoke_rowmajor_matmul(float *A, float *B, float *C, int m, int k, int n){
    dim3 blockSize(BLOCK_SIZE*BLOCK_SIZE);
    dim3 gridSize(CEIL_DIV(m, BLOCK_SIZE), CEIL_DIV(n, BLOCK_SIZE));

    myRowCoalesceKernel<32><<<gridSize, blockSize>>>(A, B, C, m, k, n);
}

int main(){
    int m = MMM;
    int n = MMN;
    int k = MMK;
    int sizeA = m * k;
    int sizeB = k * n;
    int sizeC = m * n;

    int warmup_runs = 5;
    int measurement_runs = 50;
    long long numoperations = 2LL * m * n * k;
    float *h_A, *h_B, *h_C, *h_C_cublas; //, *h_C_ref;
    float *d_A, *d_B, *d_C;
    float myelapsed_time, refelapsed_time;
    hipEvent_t mybeg, myend, refbeg, refend;

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(50.0, 25.0);

    h_A = new float[sizeA]();
    h_B = new float[sizeB]();
    h_C = new float[sizeC]();
    h_C_cublas = new float[sizeC]();
    // h_C_ref = new float[sizeC]();
    hipMalloc(&d_A, sF * sizeA);
    hipMalloc(&d_B, sF * sizeB);
    hipMalloc(&d_C, sF * sizeC);

    // float valuesA[] = {1.0f, 2.0f, 5.0f, 4.0f, 6.0f, 8.0f, 2.0f, 3.0f, 2.0f};
    populate_array(h_A, sizeA, gen, dis);
    // std::copy(valuesA,valuesA+9,h_A);
    // float valuesB[] = {1.0f, 0.0f, 2.0f, 2.0f, 1.0f, 1.0f, 8.0f, 2.0f, 4.0f};
    populate_array(h_B, sizeB, gen, dis);
    // std::copy(valuesB,valuesB+9,h_B);
    hipMemcpy(d_A, h_A, sF * sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sF * sizeB, hipMemcpyHostToDevice);
    // naive_mat_mul(h_A, h_B, h_C_ref, m, k, n);
    hipEventCreate(&mybeg);
    hipEventCreate(&myend);
    hipEventCreate(&refbeg);
    hipEventCreate(&refend);

    hipEventRecord(refbeg);
    for(int i=0; i<measurement_runs; ++i){
        run_sgemm_coalesce(m,n,k,1.0f,d_A,d_B,0.0f,d_C);
    }
    hipEventRecord(refend);
    hipEventSynchronize(refbeg);
    hipEventSynchronize(refend);
    hipEventElapsedTime(&refelapsed_time, refbeg, refend);
    std::cout<<"Ref implementation: "<<refelapsed_time<<" / "<<measurement_runs<<" = "<<refelapsed_time/measurement_runs<<"\n";

    hipEventRecord(mybeg);
    for(int i=0; i<measurement_runs; ++i){
        invoke_rowmajor_matmul(d_A, d_B, d_C, m, k, n);
    }
    hipEventRecord(myend);
    hipEventSynchronize(mybeg);
    hipEventSynchronize(myend);
    hipEventElapsedTime(&myelapsed_time, mybeg, myend);
    std::cout<<"My implementation: "<<myelapsed_time<<" / "<<measurement_runs<<" = "<<myelapsed_time/measurement_runs<<"\n";
    
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_cublas;
}
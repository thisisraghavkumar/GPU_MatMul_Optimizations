#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <random>



#define BLOCK_SIZE 32
#define MMM 1024LL
#define MMK 1024LL
#define MMN 1024LL
#define sF sizeof(float)

#define CEIL_DIV(dividend, divisor) ((dividend + divisor - 1) / divisor)

void populate_array(float *arr, int size, std::mt19937 &gen, std::uniform_real_distribution<float> &dis)
{
    for (int i = 0; i < size; i++)
    {
        arr[i] = dis(gen);
    }
}

// Kernel 2
template <const uint BLOCKSIZE>
__global__ void sgemm_global_mem_coalesce(int M, int N, int K, float alpha,
                                          const float *A, const float *B,
                                          float beta, float *C) {
  const int cRow = blockIdx.x * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
  const int cCol = blockIdx.y * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);

  // if statement is necessary to make things work under tile quantization
  if (cRow < M && cCol < N) {
    float tmp = 0.0;
    for (int i = 0; i < K; ++i) {
      tmp += A[cRow * K + i] * B[i * N + cCol];
    }
    C[cRow * N + cCol] = alpha * tmp + beta * C[cRow * N + cCol];
  }
}

void run_sgemm_coalesce(int M, int N, int K, float alpha, float *A, float *B,
                        float beta, float *C) {
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32 * 32);
  sgemm_global_mem_coalesce<32>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

template <const uint BLOCKSIZE>
__global__ void myRowCoalesceKernel(float *A, float *B, float *C, int m, int k, int n){
    int firstRow = blockIdx.x * BLOCKSIZE + threadIdx.x/BLOCKSIZE;
    int secondCol = blockIdx.y * BLOCKSIZE + threadIdx.x%BLOCKSIZE;

    if(firstRow < m && secondCol < n){
        float sum = 0.0f;
        for(int i=0; i<k; ++i){
            sum += A[firstRow * k + i] * B[i * n + secondCol];
        }
        C[firstRow * n + secondCol] = sum;
    }
}

void invoke_rowmajor_matmul(float *A, float *B, float *C, int m, int k, int n){
    dim3 blockSize(BLOCK_SIZE*BLOCK_SIZE);
    dim3 gridSize(CEIL_DIV(m, BLOCK_SIZE), CEIL_DIV(n, BLOCK_SIZE));

    myRowCoalesceKernel<32><<<gridSize, blockSize>>>(A, B, C, m, k, n);
}

// Kernel 4
template <const int BM, const int BK, const int BN, const int TM> 
__global__ void myonedtiledkernel(float *A, float *B, float *C, int m, int k, int n){
    int frameRow = blockIdx.y;
    int frameCol = blockIdx.x;

    __shared__ float As[BM*BK];
    __shared__ float Bs[BK*BN];

    int threadRow = threadIdx.x / BN;
    int threadCol = threadIdx.x % BN;
    int ARow = threadIdx.x / BK;
    int ACol = threadIdx.x % BK;
    int BRow = threadIdx.x / BN;
    int BCol = threadIdx.x % BN;

    A += (frameRow * BM * k);
    B += (frameCol * BN);
    C += (frameRow * BM * n) + (frameCol * BN);

    float results[TM] = {0.0f};
    for(int idx=0;idx<k;idx+=BK){
        As[ARow * BK + ACol] = A[ARow * k + ACol];
        Bs[BRow * BN + BCol] = B[BRow * n + BCol];
        __syncthreads();

        A += BK;
        B += (BK * n);
        for(int l=0; l<BK; l++){
            float temp = Bs[l * BN + threadCol];
            for(int i=0; i<TM; i++){
                results[i] += As[(threadRow * TM + i) * BK + l] * temp;
            }
        }
        __syncthreads();
    }
    for(int i=0; i<TM;i++){
        C[(threadRow*TM+i)*n + threadCol] = results[i];
    }
}

void invoke_oned_tiled_matmul(float *A, float *B, float *C, int m, int k, int n){
    const int BM = 64;
    const int BN = 64;
    const int BK = 8;
    const int TM = 8;
    dim3 gridDimension(CEILDIV(n, BN), CEILDIV(m, BM));
    dim3 blockDimension((BN * BM)/TM);
    myonedtiledkernel<BM,BK,BN,TM><<<gridDimension,blockDimension>>>(A, B, C, m, k, n);
}

template <const int BM, const int BN, const int BK, const int TM>
__global__ void sgemm1DBlocktiling(int M, int N, int K, float alpha,
                                   const float *A, const float *B, float beta,
                                   float *C) {
  // If we flip x and y here we get ~30% less performance for large matrices.
  // The current, 30% faster configuration ensures that blocks with sequential
  // blockIDs access columns of B sequentially, while sharing the same row of A.
  // The slower configuration would share columns of A, but access into B would
  // be non-sequential. So the faster configuration has better spatial locality
  // and hence a greater L2 hit rate.
  const uint cRow = blockIdx.y;
  const uint cCol = blockIdx.x;

  // each warp will calculate 32*TM elements, with 32 being the columnar dim.
  const int threadCol = threadIdx.x % BN;
  const int threadRow = threadIdx.x / BN;

  // allocate space for the current blocktile in SMEM
  __shared__ float As[BM * BK];
  __shared__ float Bs[BK * BN];

  // Move blocktile to beginning of A's row and B's column
  A += cRow * BM * K;
  B += cCol * BN;
  C += cRow * BM * N + cCol * BN;

  // todo: adjust this to each thread to load multiple entries and
  // better exploit the cache sizes
  assert(BM * BK == blockDim.x);
  assert(BN * BK == blockDim.x);
  const uint innerColA = threadIdx.x % BK; // warp-level GMEM coalescing
  const uint innerRowA = threadIdx.x / BK;
  const uint innerColB = threadIdx.x % BN; // warp-level GMEM coalescing
  const uint innerRowB = threadIdx.x / BN;

  // allocate thread-local cache for results in registerfile
  float threadResults[TM] = {0.0};

  // outer loop over block tiles
  for (uint bkIdx = 0; bkIdx < K; bkIdx += BK) {
    // populate the SMEM caches
    As[innerRowA * BK + innerColA] = A[innerRowA * K + innerColA];
    Bs[innerRowB * BN + innerColB] = B[innerRowB * N + innerColB];
    __syncthreads();

    // advance blocktile
    A += BK;
    B += BK * N;

    // calculate per-thread results
    for (uint dotIdx = 0; dotIdx < BK; ++dotIdx) {
      // we make the dotproduct loop the outside loop, which facilitates
      // reuse of the Bs entry, which we can cache in a tmp var.
      float tmpB = Bs[dotIdx * BN + threadCol];
      for (uint resIdx = 0; resIdx < TM; ++resIdx) {
        threadResults[resIdx] +=
            As[(threadRow * TM + resIdx) * BK + dotIdx] * tmpB;
      }
    }
    __syncthreads();
  }

  // write out the results
  for (uint resIdx = 0; resIdx < TM; ++resIdx) {
    C[(threadRow * TM + resIdx) * N + threadCol] =
        alpha * threadResults[resIdx] +
        beta * C[(threadRow * TM + resIdx) * N + threadCol];
  }
}

void runSgemm1DBlocktiling(int M, int N, int K, float alpha, float *A, float *B,
                           float beta, float *C) {
  const uint BM = 64;
  const uint BN = 64;
  const uint BK = 8;
  const uint TM = 8;
  dim3 gridDim(CEIL_DIV(N, BN), CEIL_DIV(M, BM));
  dim3 blockDim((BM * BN) / TM);
  sgemm1DBlocktiling<BM, BN, BK, TM>
      <<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

int main(){
    int m = MMM;
    int n = MMN;
    int k = MMK;
    int sizeA = m * k;
    int sizeB = k * n;
    int sizeC = m * n;

    int warmup_runs = 5;
    int measurement_runs = 50;
    long long numoperations = 2LL * m * n * k;
    float *h_A, *h_B, *h_C, *h_C_cublas; //, *h_C_ref;
    float *d_A, *d_B, *d_C;
    float myelapsed_time, refelapsed_time;
    hipEvent_t mybeg, myend, refbeg, refend;

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dis(50.0, 25.0);

    h_A = new float[sizeA]();
    h_B = new float[sizeB]();
    h_C = new float[sizeC]();
    h_C_cublas = new float[sizeC]();
    // h_C_ref = new float[sizeC]();
    hipMalloc(&d_A, sF * sizeA);
    hipMalloc(&d_B, sF * sizeB);
    hipMalloc(&d_C, sF * sizeC);

    // float valuesA[] = {1.0f, 2.0f, 5.0f, 4.0f, 6.0f, 8.0f, 2.0f, 3.0f, 2.0f};
    populate_array(h_A, sizeA, gen, dis);
    // std::copy(valuesA,valuesA+9,h_A);
    // float valuesB[] = {1.0f, 0.0f, 2.0f, 2.0f, 1.0f, 1.0f, 8.0f, 2.0f, 4.0f};
    populate_array(h_B, sizeB, gen, dis);
    // std::copy(valuesB,valuesB+9,h_B);
    hipMemcpy(d_A, h_A, sF * sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sF * sizeB, hipMemcpyHostToDevice);
    // naive_mat_mul(h_A, h_B, h_C_ref, m, k, n);
    hipEventCreate(&mybeg);
    hipEventCreate(&myend);
    hipEventCreate(&refbeg);
    hipEventCreate(&refend);

    // Calling kernel 2
    invoke_rowmajor_matmul(d_A, d_B, d_C, m, k, n);
    run_sgemm_coalesce(m,n,k,1.0f,d_A,d_B,0.0f,d_C);
    cout<<"Kernel 2 : Global coalesced memory\n";
    hipEventRecord(refbeg);
    for(int i=0; i<measurement_runs; ++i){
        run_sgemm_coalesce(m,n,k,1.0f,d_A,d_B,0.0f,d_C);
    }
    hipEventRecord(refend);
    hipEventSynchronize(refbeg);
    hipEventSynchronize(refend);
    hipEventElapsedTime(&refelapsed_time, refbeg, refend);
    std::cout<<"Ref implementation: "<<refelapsed_time<<" / "<<measurement_runs<<" = "<<refelapsed_time/measurement_runs<<"\n";

    hipEventRecord(mybeg);
    for(int i=0; i<measurement_runs; ++i){
        invoke_rowmajor_matmul(d_A, d_B, d_C, m, k, n);
    }
    hipEventRecord(myend);
    hipEventSynchronize(mybeg);
    hipEventSynchronize(myend);
    hipEventElapsedTime(&myelapsed_time, mybeg, myend);
    std::cout<<"My implementation: "<<myelapsed_time<<" / "<<measurement_runs<<" = "<<myelapsed_time/measurement_runs<<"\n";
    
    // Calling kernel 4
    runSgemm1DBlocktiling(m,n,k,1.0f,d_A,d_B,0.0f,d_C);
    invoke_oned_tiled_matmul(d_A, d_B, d_C, m, k, n);
    cout<<"Kernel 4 : 1D Block tiling\n";
    hipEventRecord(refbeg);
    for(int i=0; i<measurement_runs; ++i){
        runSgemm1DBlocktiling(m,n,k,1.0f,d_A,d_B,0.0f,d_C);
    }
    hipEventRecord(refend);
    hipEventSynchronize(refbeg);
    hipEventSynchronize(refend);
    hipEventElapsedTime(&refelapsed_time, refbeg, refend);
    std::cout<<"Ref implementation: "<<refelapsed_time<<" / "<<measurement_runs<<" = "<<refelapsed_time/measurement_runs<<"\n";

    hipEventRecord(mybeg);
    for(int i=0; i<measurement_runs; ++i){
        invoke_oned_tiled_matmul(d_A, d_B, d_C, m, k, n);
    }
    hipEventRecord(myend);
    hipEventSynchronize(mybeg);
    hipEventSynchronize(myend);
    hipEventElapsedTime(&myelapsed_time, mybeg, myend);
    std::cout<<"My implementation: "<<myelapsed_time<<" / "<<measurement_runs<<" = "<<myelapsed_time/measurement_runs<<"\n";

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_cublas;
}
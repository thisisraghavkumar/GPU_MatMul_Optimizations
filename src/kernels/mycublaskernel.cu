#include "mykernels.cuh"

/**
 * Output in C is stored in column major format.
 */
void invoke_cublas_kernel(float *A, float *B, float *C, int m, int k, int n, hipblasHandle_t &handle){
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, m, k, &alpha, B, k, A, n, &beta, C, n);
}

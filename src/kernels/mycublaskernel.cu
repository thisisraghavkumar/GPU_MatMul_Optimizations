#include "mykernels.cuh"

/**
 * Output in C is stored in column major format.
 */
void invoke_cublas_kernel(float *A, float *B, float *C, int m, int k, int n, hipblasHandle_t &handle){
    float alpha = 1.0f;
    float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, B, n, A, k, &beta, C, n);
}
